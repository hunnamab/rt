#include "hip/hip_runtime.h"
#include "rt.cuh"

__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

__global__ void kernel_getray(float3 *viewport, float3 camera, float3 *ray_buf)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    ray_buf[i].x = viewport[i].x - camera.x;
    ray_buf[i].y = viewport[i].y - camera.y;
    ray_buf[i].z = viewport[i].z - camera.z; 
}