#include "hip/hip_runtime.h"
#include "rt.cuh"

__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

__global__ void kernel_getray(float3 *viewport, float3 camera, float3 *ray_buf)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    ray_buf[i].x = viewport[i].x - camera.x;
    ray_buf[i].y = viewport[i].y - camera.y;
    ray_buf[i].z = viewport[i].z - camera.z; 
}

 __global__ void kernel_get_intersection_point(float3 *intersection_buf, float3 *ray_buf, float *depth_buf, float3 camera, int *index_buf)
 {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (index_buf[i] != -1)
    {
        intersection_buf[i] = scale(ray_buf[i], depth_buf[i]);
            intersection_buf[i] = add(\
            intersection_buf[i], camera);
    }
    else
    {
        intersection_buf[i].x = 0;
        intersection_buf[i].y = 0;
        intersection_buf[i].z = 0;
    }
}