#include "hip/hip_runtime.h"
#include "rt_device.cuh"

__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

__global__ void kernel_getray(float3 *viewport, float3 camera, float3 *ray_buf)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    ray_buf[i].x = viewport[i].x - camera.x;
    ray_buf[i].y = viewport[i].y - camera.y;
    ray_buf[i].z = viewport[i].z - camera.z; 
}

 __global__ void kernel_get_intersection_point(float3 *intersection_buf, float3 *ray_buf, float *depth_buf, float3 camera, int *index_buf)
 {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (index_buf[i] != -1)
    {
        intersection_buf[i] = scale(ray_buf[i], depth_buf[i]);
            intersection_buf[i] = add(\
            intersection_buf[i], camera);
    }
    else
    {
        intersection_buf[i].x = 0;
        intersection_buf[i].y = 0;
        intersection_buf[i].z = 0;
    }
}

__global__  void get_material_buf_device(int *index_buf, t_material *material_buf, t_object_d *objs)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = index_buf[i];
    if (j != -1)
    {
        material_buf[i].color.red = objs[j].color.red;
        material_buf[i].color.blue = objs[j].color.blue;
        material_buf[i].color.green = objs[j].color.green;
        material_buf[i].color.alpha = 255;
        material_buf[i].specular = objs[j].specular;
    }
    else
    {
        material_buf[i].color.red = 0;
        material_buf[i].color.blue = 0;
        material_buf[i].color.green = 0;
        material_buf[i].color.alpha = 255;
        material_buf[i].specular = -1;
    }
}

 __global__ void	get_frame_buf_device(t_color *frame_buf, float3	*ray_buf, float3 *intersection_buf,
	int *index_buf,
	float3 *normal_buf,
	t_material *material_buf,
	t_object_d *objs, t_light *light, int light_nmb)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = index_buf[i];
    if (j != -1)
        frame_buf[i] = reflection_color(ray_buf, normal_buf, material_buf, light, intersection_buf, objs, i, light_nmb);
    else
    {
        frame_buf[i].red = 0;
        frame_buf[i].blue = 0;
        frame_buf[i].green = 0;
        frame_buf[i].alpha = 0;
    } 
}