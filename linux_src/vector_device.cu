#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }

__device__ float	dot(float3 v1, float3 v2)
{
	return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3	sub(float3 v1, float3 v2)
{
    float3 result;

    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
	return (result);
}

__device__ float	length(float3 vector)
{
	float	res;

	res = dot(vector, vector);
	return (sqrt(res));
}

__device__ float3	div_by_scalar(float3 vector, float scalar)
{
	float3 result;

	result.x = vector.x / scalar;
	result.y = vector.y / scalar;
	result.z = vector.z / scalar;
	return (result);
}

 __device__ float3		scale(float3 v, float c)
{
	float3 res;

	res.x = v.x * c;
	res.y = v.y * c;
	res.z = v.z * c;
	return (res);
}

__device__ float3		add(const float3 v1, const float3 v2)
{
	float3 res;

	res.x = v1.x + v2.x;
	res.y = v1.y + v2.y;
	res.z = v1.z + v2.z;
	return (res);
}

__device__ float3	cross(float3 v1, float3 v2)
{
	float3 result;

	result.x = v1.y * v2.z - v1.z * v2.y;
	result.y = v1.z * v2.x - v1.x * v2.z;
	result.z = v1.x * v2.y - v1.y * v2.x;
	return (result);
}