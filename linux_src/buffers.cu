#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   buffers.c                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: pmetron <pmetron@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2020/11/07 15:38:29 by hunnamab          #+#    #+#             */
/*   Updated: 2020/11/10 17:28:32 by pmetron          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */
extern "C"{
#include "rt_cuda.h"
}
#include "rt.cuh"

void	check_funk(float3 *ray_arr, float3 *camera_start, float3 *viewport, int count)
{
	int x = 0;
	while(x < HEI * WID)
	{
		ray_arr[x].x = viewport[x].x - camera_start[0].x;
		ray_arr[x].y = viewport[x].y - camera_start[0].y;
		ray_arr[x].z = viewport[x].z - camera_start[0].z; 
		x++;
	}
}

__host__ void	get_rays_arr(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
    blockSize = 1024;

	if((hipMemcpy(scene->device_data->viewport, scene->viewport, sizeof(float3) * WID * HEI, hipMemcpyHostToDevice)) == hipSuccess)
		printf("copy to device success\n");
	kernel_getray<<<gridSize, blockSize>>>(scene->device_data->viewport, scene->camera.position, scene->device_data->ray_buf);
	if((hipMemcpy(scene->ray_buf, scene->device_data->ray_buf, sizeof(float3) * WID * HEI, hipMemcpyDeviceToHost)) == hipSuccess)
		printf("copy from device success\n");
}

void	get_closest_points(t_scene *scene, float t)
{
/* 	int x = -1;
	int i = 0;
	while(++x < WID * HEI)
	{
		t = 0;
		i = -1;
		scene->index_buf[x] = - 1;
		scene->depth_buf[x] = 100000000;
		while (++i < scene->obj_nmb)
		{
			t = scene->objs[i]->intersect(scene, i, &scene->camera.position, &scene->ray_buf[x]);
			if (t < scene->depth_buf[x] && t != 0)
			{
				scene->depth_buf[x] = t;
				scene->index_buf[x] = i;
			}
		}
	} */
	int x = 0;
	scene->objs[0]->intersect(scene, 0, &scene->camera.position, &scene->ray_buf[x]);
	x = -1;
	while(x < (WID * HEI))
	{
		scene->index_buf[x] = 0;
		x++;
	}
	x = -1;
	while(++x < WID * HEI)
	{
		if (scene->depth_buf[x])
			scene->index_buf[x] = 0;
		else
		{
			scene->index_buf[x] = -1;
		}
	}
}

void	get_intersection_buf(t_scene *scene)
{
	int x;
	int y;
	int i;

	x = 0;
	y = 0;
	while (y < HEI)
	{
		while (x < WID)
		{
			i = y * WID + x;
			if (scene->index_buf[i] != -1)
			{
				scene->intersection_buf[i] = vector_scale(\
				&scene->ray_buf[i], scene->depth_buf[i]);
				scene->intersection_buf[i] = vector_add(\
				&scene->intersection_buf[i], &scene->camera.position);
			}
			else
				scene->intersection_buf[i] = get_point(0, 0, 0);
			x++;
		}
		x = 0;
		y++;
	}
}

void	get_normal_buf(t_scene *scene)
{
	int x;
	int y;
	int i;
	int j;

	j = 0;
	x = 0;
	y = 0;
	while (y < HEI)
	{
		while (x < WID)
		{
			i = y * WID + x;
			if (scene->index_buf[i] != -1)
			{
				j = scene->index_buf[i];
				scene->objs[j]->get_normal(scene, i, j);
			}
			else
				scene->normal_buf[j] = get_point(0, 0, 0);
			x++;
		}
		x = 0;
		y++;
	}
}

void	get_material_buf(t_scene *scene)
{
	int x;
	int y;
	int i;

	y = -1;
	while (++y < HEI)
	{
		x = -1;
		while (++x < WID)
		{
			i = y * WID + x;
			if (scene->index_buf[i] != -1)
			{
				copy_color(&scene->material_buf[i].color, \
							&scene->objs[scene->index_buf[i]]->color);
				scene->material_buf[i].specular = \
				scene->objs[scene->index_buf[i]]->specular;
			}
			else
			{
				set_color_zero(&scene->material_buf[i].color);
				scene->material_buf[i].specular = -1;
			}
		}
	}
}
