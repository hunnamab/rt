#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   buffers.c                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: pmetron <pmetron@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2020/11/07 15:38:29 by hunnamab          #+#    #+#             */
/*   Updated: 2020/11/10 17:28:32 by pmetron          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */
extern "C"{
#include "rt_cuda.h"
}
#include "rt.cuh"

__host__ void	get_rays_arr(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
    blockSize = 1024;
	if((hipMemcpy(scene->device_data->viewport, scene->viewport, sizeof(float3) * WID * HEI, hipMemcpyHostToDevice)) == hipSuccess)
		printf("copy to device success\n");
	kernel_getray<<<gridSize, blockSize>>>(scene->device_data->viewport, scene->camera.position, scene->device_data->ray_buf);
	if((hipMemcpy(scene->ray_buf, scene->device_data->ray_buf, sizeof(float3) * WID * HEI, hipMemcpyDeviceToHost)) == hipSuccess)
		printf("copy from device success\n");
}

void	get_closest_points(t_scene *scene, float t)
{
	int x = -1;
	int i = 0;
	while(++x < WID * HEI)
	{
		scene->index_buf[x] = -1;
		scene->depth_buf[x] = 100000000;
	}
	hipMemcpy(scene->device_data->index_buf, scene->index_buf, sizeof(int) * WID * HEI, hipMemcpyHostToDevice);
	hipMemcpy(scene->device_data->depth_buf, scene->depth_buf, sizeof(float) * WID * HEI, hipMemcpyHostToDevice);
	i = 0;
	while(i < scene->obj_nmb)
	{
		scene->objs[i]->intersect(scene, i, &scene->camera.position, &scene->ray_buf[x]);
		i++;
	}
	hipMemcpy(scene->index_buf, scene->device_data->index_buf, sizeof(int) * WID * HEI, hipMemcpyDeviceToHost);
	hipMemcpy(scene->depth_buf, scene->device_data->depth_buf, sizeof(float) * WID * HEI, hipMemcpyDeviceToHost);
}

void	get_intersection_buf(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	kernel_get_intersection_point<<<gridSize, blockSize>>>(scene->device_data->intersection_buf,scene->device_data->ray_buf,scene->device_data->depth_buf, scene->camera.position, scene->device_data->index_buf);
	hipMemcpy(scene->intersection_buf, scene->device_data->intersection_buf, sizeof(float3) * WID * HEI, hipMemcpyDeviceToHost);
}

void	get_normal_buf(t_scene *scene)
{
/* 	int x;
	int y;
	int i;
	int j;

	j = 0;
	x = 0;
	y = 0;
	while (y < HEI)
	{
		while (x < WID)
		{
			i = y * WID + x;
			if (scene->index_buf[i] != -1)
			{
				j = scene->index_buf[i];
				scene->objs[j]->get_normal(scene, i, j);
			}
			else
				scene->normal_buf[j] = get_point(0, 0, 0);
			x++;
		}
		x = 0;
		y++;
	} */
	hipMalloc(&scene->device_data->obj, sizeof(t_object_d) * scene->obj_nmb);
	t_object_d *buf;
	buf = (t_object_d *)malloc(sizeof(t_object_d) * scene->obj_nmb);
	int i = 0;
	while(i < scene->obj_nmb)
	{
		/* if(scene->objs[i]->type == SPHERE) */
		t_sphere *s;
		s = reinterpret_cast<t_sphere *>(scene->objs[i]->data);
		buf->sphere.center = s->center;
		buf->sphere.radius = s->radius;
		i++;
	}
	hipMemcpy(scene->device_data->obj, buf, sizeof(t_object_d) * scene->obj_nmb, hipMemcpyHostToDevice);
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	get_normal_buf_device<<<gridSize, blockSize>>>(scene->device_data->obj, scene->device_data->ray_buf, scene->device_data->index_buf, scene->device_data->normal_buf, scene->device_data->intersection_buf); 
	hipMemcpy(scene->normal_buf, scene->device_data->normal_buf, sizeof(float3) * WID * HEI, hipMemcpyDeviceToHost);
}

void	get_material_buf(t_scene *scene)
{
	int x;
	int y;
	int i;

	y = -1;
	while (++y < HEI)
	{
		x = -1;
		while (++x < WID)
		{
			i = y * WID + x;
			if (scene->index_buf[i] != -1)
			{
				copy_color(&scene->material_buf[i].color, \
							&scene->objs[scene->index_buf[i]]->color);
				scene->material_buf[i].specular = \
				scene->objs[scene->index_buf[i]]->specular;
			}
			else
			{
				set_color_zero(&scene->material_buf[i].color);
				scene->material_buf[i].specular = -1;
			}
		}
	}
}
