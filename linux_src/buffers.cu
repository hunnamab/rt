#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   buffers.c                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: pmetron <pmetron@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2020/11/07 15:38:29 by hunnamab          #+#    #+#             */
/*   Updated: 2020/11/10 17:28:32 by pmetron          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */
extern "C"{
#include "rt_host.h"
}
#include "rt_device.cuh"

__host__ void	get_rays_arr(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
    blockSize = 1024;
	if((hipMemcpy(scene->device_data->viewport, scene->viewport, sizeof(float3) * WID * HEI, hipMemcpyHostToDevice)) == hipSuccess)
		printf("copy to device success\n");
	kernel_getray<<<gridSize, blockSize>>>(scene->device_data->viewport, scene->camera.position, scene->device_data->ray_buf);
	if((hipMemcpy(scene->ray_buf, scene->device_data->ray_buf, sizeof(float3) * WID * HEI, hipMemcpyDeviceToHost)) == hipSuccess)
		printf("copy from device success\n");
}

void	get_closest_points(t_scene *scene, float t)
{
	int x = -1;
	int i = 0;
	while(++x < WID * HEI)
	{
		scene->index_buf[x] = -1;
		scene->depth_buf[x] = 100000000;
	}
	hipMemcpy(scene->device_data->index_buf, scene->index_buf, sizeof(int) * WID * HEI, hipMemcpyHostToDevice);
	hipMemcpy(scene->device_data->depth_buf, scene->depth_buf, sizeof(float) * WID * HEI, hipMemcpyHostToDevice);
	i = 0;
	while(i < scene->obj_nmb)
	{
		scene->objs[i]->intersect(scene, i);
		i++;
	}
	if(scene->mode == 2)
	{
		hipMemcpy(scene->index_buf, scene->device_data->index_buf, sizeof(int) * WID * HEI, hipMemcpyDeviceToHost);
		hipMemcpy(scene->depth_buf, scene->device_data->depth_buf, sizeof(float) * WID * HEI, hipMemcpyDeviceToHost);
	}
}

void	get_intersection_buf(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	kernel_get_intersection_point<<<gridSize, blockSize>>>(scene->device_data->intersection_buf,scene->device_data->ray_buf,scene->device_data->depth_buf, scene->camera.position, scene->device_data->index_buf);
}

void	get_normal_buf(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	get_normal_buf_device<<<gridSize, blockSize>>>(scene->device_data->obj, scene->device_data->ray_buf, scene->device_data->index_buf, scene->device_data->normal_buf, scene->device_data->intersection_buf, scene->device_data->depth_buf, scene->camera.position); 
}

void	get_material_buf(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;
	int err;
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	get_material_buf_device<<<gridSize, blockSize>>>(scene->device_data->index_buf, scene->device_data->material_buf, scene->device_data->obj);
	if(scene->mode == 3)
	{
		hipMemcpy(scene->index_buf, scene->device_data->index_buf, sizeof(int) * WID * HEI, hipMemcpyDeviceToHost);
		hipMemcpy(scene->material_buf, scene->device_data->material_buf, sizeof(t_material) * WID * HEI, hipMemcpyDeviceToHost);
	}
}

 void	get_frame_buf(t_scene *scene)
{
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	get_frame_buf_device<<<gridSize, blockSize>>>(scene->device_data->frame_buf, scene->device_data->ray_buf, scene->device_data->intersection_buf,scene->device_data->index_buf, scene->device_data->normal_buf, scene->device_data->material_buf, scene->device_data->obj, scene->device_data->light, scene->light_nmb);
	hipMemcpy(scene->frame_buf, scene->device_data->frame_buf, sizeof(t_color) * WID * HEI, hipMemcpyDeviceToHost);
}
