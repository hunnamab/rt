#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   light.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: pmetron <pmetron@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2020/11/07 14:23:24 by pmetron           #+#    #+#             */
/*   Updated: 2020/11/11 12:32:22 by pmetron          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C"{
	#include "rt_host.h"
	}
	#include "rt_device.cuh"

__device__ t_color	color_mul_by_scalar(t_color *color, float scalar)
{
	t_color result_color;

	result_color.red = (float)color->red * scalar;
	result_color.blue = (float)color->blue * scalar;
	result_color.green = (float)color->green * scalar;
	return (result_color);
}


t_light		*new_light(float3 *pos_dir, int type, float intensity)
{
	t_light *new_light;

	new_light = (t_light *)protected_malloc(sizeof(t_light), 1);
	new_light->type = type;
	new_light->intensity = intensity;
	new_light->position = pos_dir[0];
	new_light->direction = pos_dir[1];
	return (new_light);
}

__device__ float3	get_light_vec(t_light *light, float3 *intersection_buf, int index, int j)
{
	float3 light_vec;

	light_vec.x = 0;
	light_vec.y = 0;
	light_vec.z = 0;
	if (light[j].type == POINT)
		light_vec = sub(light[j].position, \
		intersection_buf[index]);
	if (light[j].type == DIRECTIONAL)
		light_vec = light[j].direction;
	return (light_vec);
}

__device__ int			in_shadow(int index, float3 l)
{
/* 	float3	ray_dir;
	float3	ray_start;
	int		i;
	float	t;

	i = 0;
	ray_dir = l;
	ray_start = scale(&ray_dir, 0.0001);
	ray_start = add(&ray_start, &scene->intersection_buf[index]);
	while (i < scene->obj_nmb)
	{
		t = scene->objs[i]->intersect(scene, i, &ray_start, &ray_dir);
		if (t < 1 && t > 0.0001)
			break ;
		i++;
	}
	if (t < 1 && t > 0.0001)
		return (1); */
	return (0);
}

__device__ float		get_specular(float3 *normal_buf, float3 *ray_buf, t_material *material_buf, t_light *light, int index, int j, float3 l)
{
 	float		nri[3];
	float3		r;
	float3		d;
	float3		lb;

	lb = div_by_scalar(l, length(l));
	nri[2] = 0;
	nri[0] = dot(normal_buf[index], lb);
	r = scale(normal_buf[index], 2.0);
	r = scale(r, nri[0]);
	r = sub(r, lb);
	d.x = -ray_buf[index].x;
	d.y = -ray_buf[index].y;
	d.z = -ray_buf[index].z;
	nri[1] = dot(r, d);
	if (nri[1] > 0)
		nri[2] += light[j].intensity * pow((float)nri[1] / \
		(length(r) * length(d)), \
		material_buf[index].specular);
	return (nri[2]); 
}

__device__ t_color		reflection_color(float3 *ray_buf, float3 *normal_buf, t_material *material_buf, t_light *light, float3 *intersection_buf, t_object_d *objs, int index, int light_nmb)
{
 	float	i;
	float3	l;
	float	n_dot_l;
	int		j;

	j = -1;
	i = 0;
	while (++j < light_nmb)
	{
		if (light[j].type == AMBIENT)
			i += light[j].intensity;
		else
		{
			l = get_light_vec(light, intersection_buf, index, j);
			n_dot_l = dot(normal_buf[index], l);
			if (!(in_shadow(index, l)) && n_dot_l > 0)
			{
				if (material_buf[index].specular != -1)
					i += get_specular(normal_buf, ray_buf, material_buf, light, index, j, l);
				i += (light[j].intensity * n_dot_l) / length(l);
			}
		}
	}
	i = i > 1 ? 1 : i; 
	return (color_mul_by_scalar(&material_buf[index].color, i));
}
