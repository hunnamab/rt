#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_host.h"
    }
    #include "rt_device.cuh"

__global__  void    intersect_ray_ellipsoid_c(float3 *ray_arr, float3 camera_start, float3 e_center, float a, float b, float c, float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float k1;
    float k2;
    float k3;
    float3 dir;
    float a2;
    float b2;
    float c2;

    a2 = a * a;
    b2 = b * b;
    c2 = c * c;
    dir = ray_arr[i];
    k1 = dir.x * dir.x * b2 * c2;
    k1 += dir.y * dir.y * a2 * c2; 
    k1 += dir.z * dir.z * a2 * b2;
    k2 = camera_start.x * dir.x * b2 * c2 * 2;
    k2 += camera_start.y * dir.y * a2 * c2 * 2;
    k2 += camera_start.z * dir.z * b2 * a2 * 2;
    k3 = camera_start.x * camera_start.x * b2 * c2;
    k3 += camera_start.z * camera_start.z * a2 * b2;
    k3 += camera_start.y * camera_start.y * a2 * c2;
    k3 -= a2 * b2 * c2;
    float d = k2 * k2 - 4 * k1 * k3;
    if (d >= 0)
    {
        float t1 = (-k2 + sqrt(d)) / (2 * k1);
        float t2 = (-k2 - sqrt(d)) / (2 * k1);
        float result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}

__global__  void    intersect_ray_hyperboloid_c(float3 *ray_arr, float3 camera_start, float3 e_center, float a, float b, float c, float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float k1;
    float k2;
    float k3;
    float3 dir;
    float a2;
    float b2;
    float c2;

    a2 = a * a;
    b2 = b * b;
    c2 = c * c;
    dir = ray_arr[i];
    k1 = dir.x * dir.x * b2 * c2;
    k1 += dir.y * dir.y * a2 * c2; 
    k1 -= dir.z * dir.z * a2 * b2;
    k2 = camera_start.x * dir.x * b2 * c2 * 2;
    k2 += camera_start.y * dir.y * a2 * c2 * 2;
    k2 -= camera_start.z * dir.z * b2 * a2 * 2;
    k3 = camera_start.x * camera_start.x * b2 * c2;
    k3 -= camera_start.z * camera_start.z * a2 * b2;
    k3 += camera_start.y * camera_start.y * a2 * c2;
    k3 -= a2 * b2 * c2;
    float d = k2 * k2 - 4 * k1 * k3;
    if (d >= 0)
    {
        float t1 = (-k2 + sqrt(d)) / (2 * k1);
        float t2 = (-k2 - sqrt(d)) / (2 * k1);
        float result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}

__global__  void    intersect_ray_paraboloid_c(float3 *ray_arr, float3 camera_start, float3 e_center,float p, float q, float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float k1;
    float k2;
    float k3;
    float3 dir;

    dir = ray_arr[i];
    k1 = dir.x * dir.x + dir.y * dir.y;
    k2 = dir.x * camera_start.x + camera_start.y * dir.y - 2 * dir.z * p * q;
    k3 = q * camera_start.x * camera_start.x + p * camera_start.y * camera_start.y - 2 * camera_start.z * p * q;
    float d = k2 * k2 - 4 * k1 * k3;
    if (d >= 0)
    {
        float t1 = (-k2 + sqrt(d)) / (2 * k1);
        float t2 = (-k2 - sqrt(d)) / (2 * k1);
        float result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}

__global__ void intersect_ray_sphere_c(float3 *ray_arr, \
     float3 camera_start, \
     float3 s_center, \
     float s_radius, \
     float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = dot(ray_arr[i], ray_arr[i]);
    float b;
    float c;
    float t1;
    float t2;
    float3 dist = sub(camera_start, s_center);
    b = 2 * dot(dist, ray_arr[i]);
    c = dot(dist, dist) - (s_radius * s_radius);
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}
__global__ void intersect_ray_triangle_c(float3 *ray_arr, float3 camera_start,\
    float *depth_buf, float3 *vertex, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float3 edge[2];
    float3 vec[3];
    float det;
    float uv[2];

    edge[0] = sub(vertex[1],vertex[0]);
    edge[1] = sub(vertex[2],vertex[0]);
    vec[0] = cross(ray_arr[i], edge[1]);
    det = dot(edge[0], vec[0]);
    if (det < 1e-8 && det > -1e-8)
        return ;
    det = 1 / det;
    vec[1] = sub(camera_start,vertex[0]);
    uv[0] = dot(vec[1], vec[0]) * det;
    if (uv[0] < 0 || uv[0] > 1)
        return ;
    vec[2] = cross(vec[1], edge[0]);
    uv[1] = dot(ray_arr[i], vec[2]) * det;
    if (uv[1] < 0 || uv[0] + uv[1] > 1)
        return ;
    float res;
    res = dot(edge[1], vec[2]) * det;
    if (res > 0 && res < depth_buf[i])
    {
        depth_buf[i] = res;
        index_buf[i] = index;
        return ;
    }
}

__global__ void intersect_ray_cone_c(float3 *ray_arr, float3 camera_start, float3 position, \
    float3 vector, float angle, float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float t1;
    float t2;
    float b;
    float c;
    float ang = angle;
    float3 vec = vector;
    float3 dist = sub(camera_start, position);
    float a = dot(ray_arr[i], vec);
    a = dot(ray_arr[i], ray_arr[i]) - (1 + ang * ang) * a * a;
    b = 2 * (dot(ray_arr[i], dist) - (1 + ang * ang) * \
    dot(ray_arr[i], vec) * dot(dist, vec));
    c = dot(dist, vec);
    c = dot(dist, dist) - (1 + ang * ang) * c * c;
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}

__global__ void intersect_ray_plane_c(float3 *ray_arr, \
     float3 camera_start, \
     float *depth_buf, \
     float3 normal, \
     float d, \
     int *index_buf, \
     int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float k1;
    float k2;

    if ((dot(ray_arr[i], normal)) == 0)
        return ;
    k1 = dot(camera_start, normal) + d;
    k2 = dot(ray_arr[i], normal);
    if (k1 == 0 || (k1 < 0 && k2 < 0) || (k1 > 0 && k2 > 0))
        return ;
    k1 = -k1 / k2;
    if(k1 < depth_buf[i] && k1 > 0)
    {
        depth_buf[i] = -k1 / k2;
        index_buf[i] = index;
    }
}

__global__ void intersect_ray_cylinder_c(float3 *ray_arr, float3 camera_start, \
    float3 position, float *depth_buf, float3 vector, \
    float radius, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float t1;
    float t2;
    float b;
    float c;
    float3 dist = sub(camera_start, position);
    float a = dot(ray_arr[i], vector);
    a = dot(ray_arr[i], ray_arr[i]) - a * a;
    b = 2 * (dot(ray_arr[i], dist) - dot(ray_arr[i], vector) * \
    dot(dist, vector));
    c = dot(dist, vector);
    c = dot(dist, dist) - c * c - radius * radius;
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}