#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }
__device__ float	dot(float3 v1, float3 v2)
{
	return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3	sub(float3 v1, float3 v2)
{
    float3 result;

    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
	return (result);
}
__device__ float3		cross(float3 v1, float3 v2)
{
	float3 result;

	result.x = v1.y * v2.z - v1.z * v2.y;
	result.y = v1.z * v2.x - v1.x * v2.z;
	result.z = v1.x * v2.y - v1.y * v2.x;
	return (result);
}

__global__ void intersect_ray_sphere_cl(float3 *ray_arr, \
     float3 camera_start, \
     float3 s_center, \
     float s_radius, \
     float *depth_buf)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = dot(ray_arr[i], ray_arr[i]);
    float b;
    float c;
    float t1;
    float t2;
    float3 dist = sub(camera_start, s_center);
    b = 2 * dot(dist, ray_arr[i]);
    c = dot(dist, dist) - (s_radius * s_radius);
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        depth_buf[i] = t1 < t2 ? t1 : t2;
    }
/*     if(depth_buf[i] != 0)
        printf("depth_buf[%d] == %f\n", i, depth_buf[i]); */
}
__global__ void intersect_ray_triangle_cl(float3 *ray_arr, \
     float3 camera_start, \
     float *depth_buf, \
     float3 *vertex, float3 normal)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float3 edge[2];
    float3 vec[3];
    float det;
    float uv[2];
    float3 ver[3];
    ver[0] = vertex[0];
    ver[1] = vertex[1];
    ver[2] = vertex[2];

    edge[0] = sub(ver[1],ver[0]);
    edge[1] = sub(ver[2],ver[0]);
    vec[0] = cross(ray_arr[i], edge[1]);
    det = dot(edge[0], vec[0]);
    if (det < 1e-8 && det > -1e-8)
    {
    depth_buf[i] = 0;
    return ;
    }
    det = 1 / det;
    vec[1] = sub(camera_start,ver[0]);
    uv[0] = dot(vec[1], vec[0]) * det;
    if (uv[0] < 0 || uv[0] > 1)
    {
    depth_buf[i] = 0;
    return ;
    }
    vec[2] = cross(vec[1], edge[0]);
    uv[1] = dot(ray_arr[i], vec[2]) * det;
    if (uv[1] < 0 || uv[0] + uv[1] > 1)
    {
    depth_buf[i] = 0;
    return ;
    }
    float res;
    res = dot(edge[1], vec[2]) * det;
    if (res > 0)
    {
    depth_buf[i] = res;
    return ;
    }
    depth_buf[i] = 0;
}
