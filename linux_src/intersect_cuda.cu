#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }
__device__ float	dot(float3 v1, float3 v2)
{
	return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3	sub(float3 v1, float3 v2)
{
    float3 result;

    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
	return (result);
}

__device__ float3	cross(float3 v1, float3 v2)
{
	float3 result;

	result.x = v1.y * v2.z - v1.z * v2.y;
	result.y = v1.z * v2.x - v1.x * v2.z;
	result.z = v1.x * v2.y - v1.y * v2.x;
	return (result);
}

__global__ void intersect_ray_sphere_c(float3 *ray_arr, \
     float3 camera_start, \
     float3 s_center, \
     float s_radius, \
     float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = dot(ray_arr[i], ray_arr[i]);
    float b;
    float c;
    float t1;
    float t2;
    float3 dist = sub(camera_start, s_center);
    b = 2 * dot(dist, ray_arr[i]);
    c = dot(dist, dist) - (s_radius * s_radius);
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}
__global__ void intersect_ray_triangle_c(float3 *ray_arr, float3 camera_start,\
    float *depth_buf, float3 *vertex, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float3 edge[2];
    float3 vec[3];
    float det;
    float uv[2];

    edge[0] = sub(vertex[1],vertex[0]);
    edge[1] = sub(vertex[2],vertex[0]);
    vec[0] = cross(ray_arr[i], edge[1]);
    det = dot(edge[0], vec[0]);
    if (det < 1e-8 && det > -1e-8)
        return ;
    det = 1 / det;
    vec[1] = sub(camera_start,vertex[0]);
    uv[0] = dot(vec[1], vec[0]) * det;
    if (uv[0] < 0 || uv[0] > 1)
        return ;
    vec[2] = cross(vec[1], edge[0]);
    uv[1] = dot(ray_arr[i], vec[2]) * det;
    if (uv[1] < 0 || uv[0] + uv[1] > 1)
        return ;
    float res;
    res = dot(edge[1], vec[2]) * det;
    if (res > 0 && res < depth_buf[i])
    {
        depth_buf[i] = res;
        index_buf[i] = index;
        return ;
    }
}

__global__ void intersect_ray_cone_c(float3 *ray_arr, float3 camera_start, float3 position, \
    float3 vector, float angle, float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float t1;
    float t2;
    float b;
    float c;
    float ang = angle;
    float3 vec = vector;
    float3 dist = sub(camera_start, position);
    float a = dot(ray_arr[i], vec);
    a = dot(ray_arr[i], ray_arr[i]) - (1 + ang * ang) * a * a;
    b = 2 * (dot(ray_arr[i], dist) - (1 + ang * ang) * \
    dot(ray_arr[i], vec) * dot(dist, vec));
    c = dot(dist, vec);
    c = dot(dist, dist) - (1 + ang * ang) * c * c;
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}

__global__ void intersect_ray_plane_c(float3 *ray_arr, \
     float3 camera_start, \
     float *depth_buf, \
     float3 normal, \
     float d, \
     int *index_buf, \
     int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float k1;
    float k2;

    if ((dot(ray_arr[i], normal)) == 0)
        return ;
    k1 = dot(camera_start, normal) + d;
    k2 = dot(ray_arr[i], normal);
    if (k1 == 0 || (k1 < 0 && k2 < 0) || (k1 > 0 && k2 > 0))
        return ;
    k1 = -k1 / k2;
    if(k1 < depth_buf[i] && k1 > 0)
    {
        depth_buf[i] = -k1 / k2;
        index_buf[i] = index;
    }
}

__global__ void intersect_ray_cylinder_c(float3 *ray_arr, float3 camera_start, \
    float3 position, float *depth_buf, float3 vector, \
    float radius, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float t1;
    float t2;
    float b;
    float c;
    float3 dist = sub(camera_start, position);
    float a = dot(ray_arr[i], vector);
    a = dot(ray_arr[i], ray_arr[i]) - a * a;
    b = 2 * (dot(ray_arr[i], dist) - dot(ray_arr[i], vector) * \
    dot(dist, vector));
    c = dot(dist, vector);
    c = dot(dist, dist) - c * c - radius * radius;
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        float result;
        result = 0;
        if ((t1 < t2 && t1 > 0) || (t2 < 0 && t1 >= 0))
            result = t1;
        if ((t2 < t1 && t2 > 0) || (t1 < 0 && t2 >= 0))
            result = t2;
        if (t2 == t1 && t2 >= 0)
            result = t2;
        if (result > 0 && result < depth_buf[i])
        {
            depth_buf[i] = result;
            index_buf[i] = index;
        }
    }
}