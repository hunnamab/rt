#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }
__device__ float	dot(float3 v1, float3 v2)
{
	return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3	sub(float3 v1, float3 v2)
{
    float3 result;

    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
	return (result);
}
__device__ float3		cross(float3 v1, float3 v2)
{
	float3 result;

	result.x = v1.y * v2.z - v1.z * v2.y;
	result.y = v1.z * v2.x - v1.x * v2.z;
	result.z = v1.x * v2.y - v1.y * v2.x;
	return (result);
}

__global__ void intersect_ray_sphere_c(float3 *ray_arr, \
     float3 camera_start, \
     float3 s_center, \
     float s_radius, \
     float *depth_buf, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = dot(ray_arr[i], ray_arr[i]);
    float b;
    float c;
    float t1;
    float t2;
    float3 dist = sub(camera_start, s_center);
    b = 2 * dot(dist, ray_arr[i]);
    c = dot(dist, dist) - (s_radius * s_radius);
    c = b * b - 4 * a * c;
    if (c >= 0)
    {
        c = sqrt(c);
        t1 = (-b + c) / (2 * a);
        t2 = (-b - c) / (2 * a);
        depth_buf[i] = t1 < t2 ? t1 : t2;
    }
}
__global__ void intersect_ray_triangle_c(float3 *ray_arr, \
     float3 camera_start, \
     float *depth_buf, \
     float3 *vertex, int *index_buf, int index)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float3 edge[2];
    float3 vec[3];
    float det;
    float uv[2];

    edge[0] = sub(vertex[1],vertex[0]);
    edge[1] = sub(vertex[2],vertex[0]);
    vec[0] = cross(ray_arr[i], edge[1]);
    det = dot(edge[0], vec[0]);
    if (det < 1e-8 && det > -1e-8)
        return ;
    det = 1 / det;
    vec[1] = sub(camera_start,vertex[0]);
    uv[0] = dot(vec[1], vec[0]) * det;
    if (uv[0] < 0 || uv[0] > 1)
        return ;
    vec[2] = cross(vec[1], edge[0]);
    uv[1] = dot(ray_arr[i], vec[2]) * det;
    if (uv[1] < 0 || uv[0] + uv[1] > 1)
        return ;
    float res;
    res = dot(edge[1], vec[2]) * det;
    if (res > 0 && res < depth_buf[i])
    {
        depth_buf[i] = res;
        index_buf[i] = index;
        return ;
    }
}
