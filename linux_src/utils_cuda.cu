#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_cuda.h"
	}
#include "rt.cuh"
void	print_gpu_info(void)
{
	int val;
	hipDeviceProp_t *prop;

	prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&val);
	printf("device count %d\n", val);
	hipGetDevice(&val);
	printf("current dev %d\n", val);
	hipGetDeviceProperties(prop, 0);
	printf("device name %s\n", prop->name);
	printf("max threads per block %d\n", prop->maxThreadsPerBlock);
	printf("multiProcessorCount %d\n", prop->multiProcessorCount);
	printf("regsPerBlock %d\n", prop->regsPerBlock);
	printf("warp size %d\n", prop->warpSize);
	printf("canMapHostMemory %d\n", prop->canMapHostMemory);
	printf("maxGridSize[3] (%d,%d,%d)\n", prop->maxGridSize[0],prop->maxGridSize[1],prop->maxGridSize[2]);
}

__host__ void 	intersect_ray_sphere(t_scene *scene, int index, float3 *start, float3 *dir)
{
    dim3     gridSize;
    dim3     blockSize;
	t_sphere *sphere;
	gridSize = WID * HEI / 256;
	blockSize = 256;
	sphere = reinterpret_cast<t_sphere *>(scene->objs[index]->data);
    intersect_ray_sphere_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,sphere->center,sphere->radius,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_triangle(t_scene *scene, int index, float3 *start, float3 *dir)
{
	t_triangle	*t;

	t = reinterpret_cast<t_triangle *>(scene->objs[index]->data);
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 256;
	blockSize = 256;
	float3 *vertex;
	hipMalloc(&vertex, sizeof(float3) * 3);
	hipMemcpy(vertex, t->vertex,sizeof(float3) * 3, hipMemcpyHostToDevice);
    intersect_ray_triangle_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,scene->device_data->depth_buf, vertex, scene->device_data->index_buf, index);
	hipFree(vertex);
}
