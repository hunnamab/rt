#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_cuda.h"
    }
void	print_gpu_info(void)
{
	int val;
	hipDeviceProp_t *prop;

	prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&val);
	printf("device count %d\n", val);
	hipGetDevice(&val);
	printf("current dev %d\n", val);
	hipGetDeviceProperties(prop, 0);
	printf("device name %s\n", prop->name);
	printf("max threads per block %d\n", prop->maxThreadsPerBlock);
	printf("multiProcessorCount %d\n", prop->multiProcessorCount);
	printf("regsPerBlock %d\n", prop->regsPerBlock);
	printf("warp size %d\n", prop->warpSize);
	printf("canMapHostMemory %d\n", prop->canMapHostMemory);
	printf("maxGridSize[3] (%d,%d,%d)\n", prop->maxGridSize[0],prop->maxGridSize[1],prop->maxGridSize[2]);
}