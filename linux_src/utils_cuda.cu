#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_cuda.h"
	}
#include "rt.cuh"
void	print_gpu_info(void)
{
	int val;
	hipDeviceProp_t *prop;

	prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&val);
	printf("device count %d\n", val);
	hipGetDevice(&val);
	printf("current dev %d\n", val);
	hipGetDeviceProperties(prop, 0);
	printf("device name %s\n", prop->name);
	printf("max threads per block %d\n", prop->maxThreadsPerBlock);
	printf("multiProcessorCount %d\n", prop->multiProcessorCount);
	printf("regsPerBlock %d\n", prop->regsPerBlock);
	printf("warp size %d\n", prop->warpSize);
	printf("canMapHostMemory %d\n", prop->canMapHostMemory);
	printf("maxGridSize[3] (%d,%d,%d)\n", prop->maxGridSize[0],prop->maxGridSize[1],prop->maxGridSize[2]);
}

__host__ void 	intersect_ray_sphere(t_scene *scene, int index, float3 *start, float3 *dir)
{
    dim3     gridSize;
    dim3     blockSize;
	t_sphere *sphere;
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	sphere = reinterpret_cast<t_sphere *>(scene->objs[index]->data);
    intersect_ray_sphere_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,sphere->center,sphere->radius,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_triangle(t_scene *scene, int index, float3 *start, float3 *dir)
{
	t_triangle	*t;

	t = reinterpret_cast<t_triangle *>(scene->objs[index]->data);
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	float3 *vertex;
	hipMalloc(&vertex, sizeof(float3) * 3);
	hipMemcpy(vertex, t->vertex,sizeof(float3) * 3, hipMemcpyHostToDevice);
    intersect_ray_triangle_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,scene->device_data->depth_buf, vertex, scene->device_data->index_buf, index);
	hipFree(vertex);
}


__host__ void	intersect_ray_cone(t_scene *scene, int index, float3 *start, float3 *dir)
{
	t_cone *c;
	dim3     gridSize;
    dim3     blockSize;
	c = reinterpret_cast<t_cone *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_cone_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,c->position, c->vec, c->angle, scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_cylinder(t_scene *scene, int index, float3 *start, float3 *dir)
{
	t_cylinder *c;

	dim3     gridSize;
    dim3     blockSize;
	c = reinterpret_cast<t_cylinder *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_cylinder_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,c->position, scene->device_data->depth_buf, c->vec, c->radius, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_plane(t_scene *scene, int index, float3 *start, float3 *dir)
{
	t_plane *p;
	dim3     gridSize;
	dim3     blockSize;
	
	p = reinterpret_cast<t_plane *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_plane_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position, scene->device_data->depth_buf, p->normal, p->d, scene->device_data->index_buf, index);
}