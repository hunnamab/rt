#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_cuda.h"
	}
#include "rt.cuh"
void	print_gpu_info(void)
{
	int val;
	hipDeviceProp_t *prop;

	prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&val);
	printf("device count %d\n", val);
	hipGetDevice(&val);
	printf("current dev %d\n", val);
	hipGetDeviceProperties(prop, 0);
	printf("device name %s\n", prop->name);
	printf("max threads per block %d\n", prop->maxThreadsPerBlock);
	printf("multiProcessorCount %d\n", prop->multiProcessorCount);
	printf("regsPerBlock %d\n", prop->regsPerBlock);
	printf("warp size %d\n", prop->warpSize);
	printf("canMapHostMemory %d\n", prop->canMapHostMemory);
	printf("maxGridSize[3] (%d,%d,%d)\n", prop->maxGridSize[0],prop->maxGridSize[1],prop->maxGridSize[2]);
}

void prepare(t_scene *scene, t_sphere *sphere)
{
    dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 256;
	blockSize = 256;
    intersect_ray_sphere_cl<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,sphere->center,sphere->radius,scene->device_data->depth_buf);
    hipMemcpy(scene->depth_buf, scene->device_data->depth_buf, sizeof(float) * WID * HEI,hipMemcpyDeviceToHost);
}

void prepare_t(t_scene *scene, t_triangle *t)
{
    dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 256;
	blockSize = 256;
	float3 *vertex;
	hipMalloc(&vertex, sizeof(float3) * 3);
	hipMemcpy(vertex, t->vertex,sizeof(float3) * 3, hipMemcpyHostToDevice);
    intersect_ray_triangle_cl<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,scene->device_data->depth_buf,vertex, t->normal);
    hipMemcpy(scene->depth_buf, scene->device_data->depth_buf, sizeof(float) * WID * HEI,hipMemcpyDeviceToHost);
}