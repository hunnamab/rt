#include "hip/hip_runtime.h"
extern "C"{
	#include "rt_host.h"
	}
	#include "rt_device.cuh"
void	print_gpu_info(void)
{
	int val;
	hipDeviceProp_t *prop;

	prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
	hipGetDeviceCount(&val);
	printf("device count %d\n", val);
	hipGetDevice(&val);
	printf("current dev %d\n", val);
	hipGetDeviceProperties(prop, 0);
	printf("device name %s\n", prop->name);
	printf("max threads per block %d\n", prop->maxThreadsPerBlock);
	printf("multiProcessorCount %d\n", prop->multiProcessorCount);
	printf("regsPerBlock %d\n", prop->regsPerBlock);
	printf("warp size %d\n", prop->warpSize);
	printf("canMapHostMemory %d\n", prop->canMapHostMemory);
	printf("maxGridSize[3] (%d,%d,%d)\n", prop->maxGridSize[0],prop->maxGridSize[1],prop->maxGridSize[2]);
}
//тут утечка пока что
__host__ void	device_objects_init(t_scene *scene)
{
	t_object_d *buf;
	int i = 0;

	hipMalloc(&scene->device_data->obj, sizeof(t_object_d) * scene->obj_nmb);
	buf = (t_object_d *)malloc(sizeof(t_object_d) * scene->obj_nmb);
	while(i < scene->obj_nmb)
	{
		buf[i].type = scene->objs[i]->type;
		if(scene->objs[i]->type == SPHERE)
		{
			t_sphere *s;
			s = reinterpret_cast<t_sphere *>(scene->objs[i]->data);
			buf[i].sphere.center = s->center;
			buf[i].sphere.radius = s->radius;
		}
		if(scene->objs[i]->type == TRIANGLE)
		{
			t_triangle *t;
			t = reinterpret_cast<t_triangle *>(scene->objs[i]->data);
			buf[i].triangle.vertex[0] = t->vertex[0];
			buf[i].triangle.vertex[1] = t->vertex[1];
			buf[i].triangle.vertex[2] = t->vertex[2];
			buf[i].triangle.normal = t->normal;
		}
		if(scene->objs[i]->type == CONE)
		{
			t_cone *cone;
			cone = reinterpret_cast<t_cone *>(scene->objs[i]->data);
			buf[i].cone.position = cone->position;
			buf[i].cone.vec = cone->vec;
			buf[i].cone.angle = cone->angle;
		}
		if(scene->objs[i]->type == CYLINDER)
		{
			t_cylinder *cylinder;
			cylinder = reinterpret_cast<t_cylinder *>(scene->objs[i]->data);
			buf[i].cylinder.position = cylinder->position;
			buf[i].cylinder.vec = cylinder->vec;
			buf[i].cylinder.radius = cylinder->radius;
		}
		if(scene->objs[i]->type == PLANE)
		{
			t_plane *p;
			p = reinterpret_cast<t_plane *>(scene->objs[i]->data);
			buf[i].plane.normal = p->normal;
			buf[i].plane.point = p->point;
			buf[i].plane.d = p->d;
		}
		if(scene->objs[i]->type == ELLIPSOID)
		{
			t_ellipsoid *el;
			el = reinterpret_cast<t_ellipsoid *>(scene->objs[i]->data);
			buf[i].ellipsoid.a = el->a;
			buf[i].ellipsoid.b = el->b;
			buf[i].ellipsoid.c = el->c;
			buf[i].ellipsoid.center = el->center;
		}
		if(scene->objs[i]->type == HYPERBOLOID)
		{
			t_hyperboloid *hr;
			hr = reinterpret_cast<t_hyperboloid *>(scene->objs[i]->data);
			buf[i].hyperboloid.a = hr->a;
			buf[i].hyperboloid.b = hr->b;
			buf[i].hyperboloid.c = hr->c;
			buf[i].hyperboloid.center = hr->center;
		}
		if(scene->objs[i]->type == PARABOLOID)
		{
			t_paraboloid *pr;
			pr = reinterpret_cast<t_paraboloid *>(scene->objs[i]->data);
			buf[i].paraboloid.p = pr->p;
			buf[i].paraboloid.q = pr->q;
			buf[i].paraboloid.center = pr->center;
		}
		buf[i].color.red = scene->objs[i]->color.red;
		buf[i].color.blue = scene->objs[i]->color.blue;
		buf[i].color.green = scene->objs[i]->color.green;
		buf[i].color.alpha = scene->objs[i]->color.alpha;
		buf[i].specular = scene->objs[i]->specular;
		i++;
	}
	hipMemcpy(scene->device_data->obj, buf, sizeof(t_object_d) * scene->obj_nmb, hipMemcpyHostToDevice);
}

__host__ void	device_light_init(t_scene *scene)
{
	t_light *buf;
	int i = 0;
	buf = (t_light *)malloc(sizeof(t_light) * scene->light_nmb);
	hipMalloc(&scene->device_data->light, sizeof(t_light) * scene->light_nmb);
	while(i < scene->light_nmb)
	{
		buf[i].type = scene->light[i]->type;
		buf[i].intensity = scene->light[i]->intensity;
		if(scene->light[i]->type == DIRECTIONAL)
		{
			buf[i].direction = scene->light[i]->direction;
			buf[i].position = scene->light[i]->position;
		}
		if(scene->light[i]->type == POINT)
			buf[i].position = scene->light[i]->position;
		i++;
	}
	if((hipMemcpy(scene->device_data->light, buf, sizeof(t_light) * scene->light_nmb, hipMemcpyHostToDevice)) == hipSuccess)
		printf("light device mem copy success\n");
}

__host__ void 	intersect_ray_sphere(t_scene *scene, int index)
{
    dim3     gridSize;
    dim3     blockSize;
	t_sphere *sphere;
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	sphere = reinterpret_cast<t_sphere *>(scene->objs[index]->data);
    intersect_ray_sphere_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,sphere->center,sphere->radius,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_triangle(t_scene *scene, int index)
{
	t_triangle	*t;

	t = reinterpret_cast<t_triangle *>(scene->objs[index]->data);
	dim3     gridSize;
    dim3     blockSize;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	float3 *vertex;
	hipMalloc(&vertex, sizeof(float3) * 3);
	hipMemcpy(vertex, t->vertex,sizeof(float3) * 3, hipMemcpyHostToDevice);
    intersect_ray_triangle_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,scene->device_data->depth_buf, vertex, scene->device_data->index_buf, index);
	hipFree(vertex);
}


__host__ void	intersect_ray_cone(t_scene *scene, int index)
{
	t_cone *c;
	dim3     gridSize;
    dim3     blockSize;
	c = reinterpret_cast<t_cone *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_cone_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,c->position, c->vec, c->angle, scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_cylinder(t_scene *scene, int index)
{
	t_cylinder *c;

	dim3     gridSize;
    dim3     blockSize;
	c = reinterpret_cast<t_cylinder *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_cylinder_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,c->position, scene->device_data->depth_buf, c->vec, c->radius, scene->device_data->index_buf, index);
}

__host__ void	intersect_ray_plane(t_scene *scene, int index)
{
	t_plane *p;
	dim3     gridSize;
	dim3     blockSize;
	
	p = reinterpret_cast<t_plane *>(scene->objs[index]->data);
	gridSize = WID * HEI / 1024;
	blockSize = 1024;
    intersect_ray_plane_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position, scene->device_data->depth_buf, p->normal, p->d, scene->device_data->index_buf, index);
}

__host__ void 	intersect_ray_ellipsoid(t_scene *scene, int index)
{
    dim3     gridSize;
    dim3     blockSize;
	t_ellipsoid *el;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	el = reinterpret_cast<t_ellipsoid *>(scene->objs[index]->data);
    intersect_ray_ellipsoid_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,el->center,el->a, el->b,el->c,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void 	intersect_ray_hyperboloid(t_scene *scene, int index)
{
    dim3     gridSize;
    dim3     blockSize;
	t_hyperboloid *hr;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	hr = reinterpret_cast<t_hyperboloid *>(scene->objs[index]->data);
    intersect_ray_hyperboloid_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,hr->center,hr->a, hr->b,hr->c,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}

__host__ void 	intersect_ray_paraboloid(t_scene *scene, int index)
{
    dim3     gridSize;
    dim3     blockSize;
	t_paraboloid *pr;

	gridSize = WID * HEI / 1024;
	blockSize = 1024;
	pr = reinterpret_cast<t_paraboloid *>(scene->objs[index]->data);
    intersect_ray_paraboloid_c<<<gridSize,blockSize>>>(scene->device_data->ray_buf,scene->camera.position,pr->center,pr->p,pr->q,scene->device_data->depth_buf, scene->device_data->index_buf, index);
}