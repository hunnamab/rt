#include "hip/hip_runtime.h"
extern "C"{
    #include "rt_host.h"
    }
    #include "rt_device.cuh"

    __device__ void get_normal_sphere(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
    {
        float l;
        int j;
    
        j = index_buf[index];
        normal_buf[index] = sub(intersection_buf[index], obj[j].sphere.center);
        l = length(normal_buf[index]);
        normal_buf[index] = div_by_scalar(normal_buf[index], l);
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }
    __device__ void get_normal_ellipsoid(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
    {
        float l;
        int j;
    
        j = index_buf[index];
        normal_buf[index] = sub(intersection_buf[index], obj[j].ellipsoid.center);
        l = length(normal_buf[index]);
        normal_buf[index] = div_by_scalar(normal_buf[index], l);
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }
    __device__ void get_normal_plane(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
    {
        int j;
    
        j = index_buf[index];
        normal_buf[index] = obj[j].plane.normal;
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }

    __device__ void get_normal_triangle(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
    {
        int j;
    
        j = index_buf[index];
        normal_buf[index] = obj[j].triangle.normal;
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }

    __device__ void get_normal_cone(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index, float3 camera_position, float *depth_buf)
    {
        float	m;
        float3 buf;
        int j;
    
        j = index_buf[index];
        buf = sub(camera_position, obj[j].cone.position);
        m = dot(ray_buf[index],  obj[j].cone.vec) * \
                        depth_buf[index] + dot(buf, obj[j].cone.vec);
        buf = scale(obj[j].cone.vec, m);
        normal_buf[index] = scale(buf, (1 +  obj[j].cone.angle *  obj[j].cone.angle));
        buf = sub(intersection_buf[index], obj[j].cone.position);
        normal_buf[index] = sub(buf, normal_buf[index]);
        normal_buf[index] = div_by_scalar(normal_buf[index], \
                                    length(normal_buf[index]));
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }

    __device__ void get_normal_cylinder(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index, float3 camera_position, float *depth_buf)
    {
        float		m;
        float3		p;
        float3		buf[2];
        int j;
    
        j = index_buf[index];
        buf[0] = sub(camera_position, obj[j].cylinder.position);
        m = dot(ray_buf[index], obj[j].cylinder.vec) * \
            depth_buf[index] + dot(buf[0], obj[j].cylinder.vec);
        buf[0] = scale(ray_buf[index], depth_buf[index]);
        p = add(camera_position, buf[0]);
        buf[0] = sub(p, obj[j].cylinder.position);
        buf[1] = scale(obj[j].cylinder.vec, m);
        normal_buf[index] = sub(buf[0], buf[1]);
        normal_buf[index] = div_by_scalar(normal_buf[index], \
                                    length(normal_buf[index]));
        if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
            normal_buf[index] = scale(normal_buf[index], -1);
    }

    __global__ void	get_normal_buf_device(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, float *depth_buf, float3 camera_position)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = index_buf[i];
    
        if(j != -1)
        {
            if (obj[j].type == SPHERE)
                get_normal_sphere(obj, ray_buf, index_buf, normal_buf,intersection_buf, i);
            else if (obj[j].type == CONE)
                get_normal_cone(obj, ray_buf, index_buf, normal_buf,intersection_buf, i, camera_position, depth_buf);
            else if (obj[j].type == CYLINDER)
                get_normal_cylinder(obj, ray_buf, index_buf, normal_buf,intersection_buf, i, camera_position, depth_buf);
            else if (obj[j].type == PLANE)
                get_normal_plane(obj, ray_buf, index_buf, normal_buf,intersection_buf, i);
            else if (obj[j].type == TRIANGLE)
                get_normal_triangle(obj, ray_buf, index_buf, normal_buf,intersection_buf, i);
            else if (obj[j].type == ELLIPSOID)
                get_normal_ellipsoid(obj, ray_buf, index_buf, normal_buf,intersection_buf, i);
        }
    }