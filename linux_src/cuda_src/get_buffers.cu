#include "hip/hip_runtime.h"
#include "rt.cuh"

__global__ void kernel_getray(float3 *viewport, float3 *camera, float3 *ray_buf)
{
    int i = blockDim.x * blockIdx.x * threadIdx.x;

    ray_buf[i].x = viewport[i].x - camera[0].x;
    ray_buf[i].y = viewport[i].y - camera[0].y;
    ray_buf[i].z = viewport[i].z - camera[0].z;
}