#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }

__device__ void get_normal_sphere(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
{
	float l;
	int j;

	j = index_buf[index];
	normal_buf[index] = sub(intersection_buf[index], obj[j].sphere.center);
	l = length(normal_buf[index]);
	normal_buf[index] = div_by_scalar(normal_buf[index], l);
	if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
		normal_buf[index] = scale(normal_buf[index], -1);
}

//__global__ void	get_normal_buf_device(t_object_device *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf)
__global__ void	get_normal_buf_device(t_object_d *obj, float3 *ray_buf, int *index_buf, float3 *normal_buf, float3 *intersection_buf)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = index_buf[i];
/* 	int j = index_buf[i];
	
 	if(/* obj[j].type == SPHERE &&  j != -1)
 	{
		t_sphere *s;
		s = reinterpret_cast<t_sphere *>(obj[j].data);
		get_normal_sphere(s, ray_buf, index_buf, normal_buf,intersection_buf, i);
	} */
	if(/* obj[j].type == SPHERE &&*/  j != -1)
	{
		get_normal_sphere(obj,ray_buf, index_buf, normal_buf,intersection_buf, i);
	}
}

__global__ void new_sphere_device(t_object_device *obj, float3 center, float radius, t_color color, \
	float *rotation, float specular, int index)
{
	t_sphere *s;
	s = (t_sphere *)malloc(sizeof(t_sphere));
	s->center = center;
	s->radius = radius;
	obj[index].data = s;
	obj[index].color = color;
	obj[index].specular = specular;
	obj[index].rotation = (float *)malloc(sizeof(float) * 3);
	obj[index].rotation[0] = rotation[0];
	obj[index].rotation[1] = rotation[1];
	obj[index].rotation[2] = rotation[2];
}

__host__ int    cuda_init(t_scene *scene)
{
	scene->device_data = (t_scene_device *)malloc(sizeof(t_scene_device));
	print_gpu_info();
	int err = 0;
	
	/* if((hipMalloc(&scene->device_data->objs, sizeof(t_object_device) * scene->obj_nmb)) == hipSuccess)
		printf("objs_allocated on device \n"); */
	if((hipMalloc(&scene->device_data->ray_buf, sizeof(float3) * WID * HEI)) == hipSuccess)
		printf("device ray_buf allocated\n");
	else
		printf("device ray_buf alloc error %d\n", err);
	if((hipMalloc(&scene->device_data->intersection_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device intersection_buf allocated\n");
	else
		printf("device intersection_buf alloc error\n");
 	if((hipMalloc(&scene->device_data->viewport, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device viewport allocated\n");
	else
		printf("device viewport alloc error\n");
 	if((hipMalloc(&scene->device_data->index_buf, sizeof(int) * WID * HEI) == hipSuccess))
		printf("device index_buf allocated\n");
	else
		printf("device index_buf alloc error\n");
	if((hipMalloc(&scene->device_data->depth_buf, sizeof(float) * WID * HEI) == hipSuccess))
		printf("device depth_buf allocated\n");
	else
		printf("device depth_buf alloc error\n");
	if((hipMalloc(&scene->device_data->normal_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device normal_buf allocated\n");
	else
		printf("device normal_buf alloc error\n");
	if((hipMalloc(&scene->device_data->camera, sizeof(float3))) == hipSuccess)
		printf("device camera allocated\n");
	/* int i = 0;
	while(i < scene->obj_nmb)
		{
			t_sphere *s;
			s = reinterpret_cast<t_sphere *>(scene->objs[i]->data);
			float *rotation;
			hipMalloc(&rotation, sizeof(float) * 3);
			hipMemcpy(rotation, scene->objs[i]->rotation, sizeof(float) * 3, hipMemcpyHostToDevice);
			new_sphere_device<<<1,1>>>(scene->device_data->objs, s->center, s->radius, scene->objs[i]->color, rotation, scene->objs[i]->specular, i);
			hipFree(&rotation);
			i++;
		} */
	return (0);
}
