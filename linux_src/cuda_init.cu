#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }

__device__ void lol(void)
{
	printf("kek is real\n");
}
typedef void (*lol_p)(void);
__device__ lol_p pfunc1 = lol;

__global__ void  obj_init(t_object_device *objs, lol_p *kek)
{
	objs[0].data = malloc(sizeof(t_sphere));
	/* if((malloc(&objs[0].data, sizeof(t_sphere))) == hipSuccess) */
	if(objs[0].data != NULL)	
		printf("objs->data allocated on device \n");
	objs->lol_p = kek[0];
	objs->lol_p();
}
__host__ int    cuda_init(t_scene *scene)
{
	scene->device_data = (t_scene_device *)malloc(sizeof(t_scene_device));
	print_gpu_info();
	int err = 0;
	lol_p *host_lol_ptr;
	lol_p *dev_lol_ptr;
	host_lol_ptr = (lol_p *)malloc(sizeof(lol_p));
	hipMalloc(&dev_lol_ptr, sizeof(lol_p));
	hipMemcpyFromSymbol(&host_lol_ptr[0], HIP_SYMBOL(pfunc1), sizeof(lol_p));
	hipMemcpy(dev_lol_ptr, host_lol_ptr, sizeof(lol_p), hipMemcpyHostToDevice);
	if((hipMalloc(&scene->device_data->objs, sizeof(t_object_device) * scene->obj_nmb)) == hipSuccess)
		printf("objs_allocated on device \n");
	obj_init<<<1,1>>>(scene->device_data->objs, dev_lol_ptr);
	if((hipMalloc(&scene->device_data->ray_buf, sizeof(float3) * WID * HEI)) == hipSuccess)
		printf("device ray_buf allocated\n");
	else
		printf("device ray_buf alloc error %d\n", err);
	if((hipMalloc(&scene->device_data->intersection_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device intersection_buf allocated\n");
	else
		printf("device intersection_buf alloc error\n");
 	if((hipMalloc(&scene->device_data->viewport, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device viewport allocated\n");
	else
		printf("device viewport alloc error\n");
 	if((hipMalloc(&scene->device_data->index_buf, sizeof(int) * WID * HEI) == hipSuccess))
		printf("device index_buf allocated\n");
	else
		printf("device index_buf alloc error\n");
	if((hipMalloc(&scene->device_data->depth_buf, sizeof(float) * WID * HEI) == hipSuccess))
		printf("device depth_buf allocated\n");
	else
		printf("device depth_buf alloc error\n");
	if((hipMalloc(&scene->device_data->normal_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device normal_buf allocated\n");
	else
		printf("device normal_buf alloc error\n");
	if((hipMalloc(&scene->device_data->camera, sizeof(float3))) == hipSuccess)
		printf("device camera allocated\n");
	return (0);
}
