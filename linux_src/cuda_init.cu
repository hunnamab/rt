#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }

__host__ int    cuda_init(t_scene *scene)
{
	scene->device_data = (t_scene_device *)malloc(sizeof(t_scene_device));
	print_gpu_info();
	int err = 0;
	
	if((hipMalloc(&scene->device_data->ray_buf, sizeof(float3) * WID * HEI)) == hipSuccess)
		printf("device ray_buf allocated\n");
	else
		printf("device ray_buf alloc error %d\n", err);
	if((hipMalloc(&scene->device_data->intersection_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device intersection_buf allocated\n");
	else
		printf("device intersection_buf alloc error\n");
 	if((hipMalloc(&scene->device_data->viewport, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device viewport allocated\n");
	else
		printf("device viewport alloc error\n");
 	if((hipMalloc(&scene->device_data->index_buf, sizeof(int) * WID * HEI) == hipSuccess))
		printf("device index_buf allocated\n");
	else
		printf("device index_buf alloc error\n");
	if((hipMalloc(&scene->device_data->depth_buf, sizeof(float) * WID * HEI) == hipSuccess))
		printf("device depth_buf allocated\n");
	else
		printf("device depth_buf alloc error\n");
	if((hipMalloc(&scene->device_data->normal_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device normal_buf allocated\n");
	else
		printf("device normal_buf alloc error\n");
	if((hipMalloc(&scene->device_data->frame_buf, sizeof(t_color) * WID * HEI) == hipSuccess))
		printf("device normal_buf allocated\n");
	else
		printf("device normal_buf alloc error\n");
	if((hipMalloc(&scene->device_data->camera, sizeof(float3))) == hipSuccess)
		printf("device camera allocated\n");
	if((hipMalloc(&scene->device_data->material_buf, sizeof(t_material) * WID * HEI) == hipSuccess))
		printf("device material_buf allocated\n");
	else
		printf("device material_buf alloc error\n");
	device_objects_init(scene);
	device_light_init(scene);
	return (0);
}
