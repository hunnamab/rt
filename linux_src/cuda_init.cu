#include "hip/hip_runtime.h"
#include "rt.cuh"
extern "C"{
    #include "rt_cuda.h"
    }

__device__ void get_normal_sphere(t_object_device *obj, int *index_buf, float3 *normal_buf, float3 *intersection_buf, int index)
{
	t_sphere *s;
	int j;
	float l;

	j = index_buf[index];
	s = (t_sphere *)obj[j]->data;
	normal_buf[index] = sub(intersection_buf[index], s->center);
	l = length(normal_buf[index]);
	normal_buf[index] = div_by_scalar(normal_buf[index], l);
	if (dot(ray_buf[index], normal_buf[index]) > 0.0001)
		normal_buf[index] = scale(normal_buf[index], -1);
}
typedef void (*get_normal)(void);
__device__ get_normal p_get_normal_sphere = get_normal_sphere;

__global__ void	get_normal_buf_device(t_object_device *obj, int *index_buf, float3 *normal_buf, float3 *intersection_buf)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = index_buf[i];
	if (obj[j])
}

__global__ void new_sphere_device(t_object_device *obj, get_normal *get_normal, float3 center, float radius, t_color color, \
	float *rotation, float specular, int index)
{
	t_sphere *s;
	s = (t_sphere *)malloc(sizeof(t_sphere));
	s->center = center;
	s->radius = radius;
	obj[index].data = s;
	obj[index].color = color;
	obj[index].specular = specular;
	obj[index].rotation = (float *)malloc(sizeof(float) * 3);
	obj[index].rotation[0] = rotation[0];
	obj[index].rotation[1] = rotation[1];
	obj[index].rotation[2] = rotation[2];
	obj[index].get_normal = get_normal[0];
}
__global__ void test(t_object_device *objs, int index)
{
	t_sphere *s;

	s = reinterpret_cast<t_sphere *>(objs[index].data);
	printf("s->center %f\n", s->center);
	printf("s->radius %f\n", s->radius);
	printf("objs[%d].specular %f\n", objs[index].specular);
	objs[index].get_normal();
}
__host__ int    cuda_init(t_scene *scene)
{
	scene->device_data = (t_scene_device *)malloc(sizeof(t_scene_device));
	print_gpu_info();
	int err = 0;
	get_normal *host_get_normal_ptr;
	get_normal *dev_get_normal_ptr;
	host_get_normal_ptr = (get_normal *)malloc(sizeof(get_normal));
	hipMalloc(&dev_get_normal_ptr, sizeof(get_normal));
	hipMemcpyFromSymbol(&host_get_normal_ptr[0], HIP_SYMBOL(p_get_normal_sphere), sizeof(get_normal));
	hipMemcpy(dev_get_normal_ptr, host_get_normal_ptr, sizeof(get_normal), hipMemcpyHostToDevice);
	int i = 0;
	//obj_init<<<1,1>>>(scene->device_data->objs, scene->obj_nmb);
	hipMalloc(&scene->device_data->objs, sizeof(t_object_device) * scene->obj_nmb);
	while(i < scene->obj_nmb)
	{
		t_sphere *s;
		s = reinterpret_cast<t_sphere *>(scene->objs[i]->data);
		float *rotation;
		hipMalloc(&rotation, sizeof(float) * 3);
		hipMemcpy(rotation, scene->objs[i]->rotation, sizeof(float) * 3, hipMemcpyHostToDevice);
		new_sphere_device<<<1,1>>>(scene->device_data->objs,dev_get_normal_ptr, s->center, s->radius, scene->objs[i]->color, rotation, scene->objs[i]->specular, i);
		hipFree(&rotation);
		i++;
	}
	test<<<1,1>>>(scene->device_data->objs, 0);
	if((hipMalloc(&scene->device_data->objs, sizeof(t_object_device) * scene->obj_nmb)) == hipSuccess)
		printf("objs_allocated on device \n");
	if((hipMalloc(&scene->device_data->ray_buf, sizeof(float3) * WID * HEI)) == hipSuccess)
		printf("device ray_buf allocated\n");
	else
		printf("device ray_buf alloc error %d\n", err);
	if((hipMalloc(&scene->device_data->intersection_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device intersection_buf allocated\n");
	else
		printf("device intersection_buf alloc error\n");
 	if((hipMalloc(&scene->device_data->viewport, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device viewport allocated\n");
	else
		printf("device viewport alloc error\n");
 	if((hipMalloc(&scene->device_data->index_buf, sizeof(int) * WID * HEI) == hipSuccess))
		printf("device index_buf allocated\n");
	else
		printf("device index_buf alloc error\n");
	if((hipMalloc(&scene->device_data->depth_buf, sizeof(float) * WID * HEI) == hipSuccess))
		printf("device depth_buf allocated\n");
	else
		printf("device depth_buf alloc error\n");
	if((hipMalloc(&scene->device_data->normal_buf, sizeof(float3) * WID * HEI) == hipSuccess))
		printf("device normal_buf allocated\n");
	else
		printf("device normal_buf alloc error\n");
	if((hipMalloc(&scene->device_data->camera, sizeof(float3))) == hipSuccess)
		printf("device camera allocated\n");
	return (0);
}
